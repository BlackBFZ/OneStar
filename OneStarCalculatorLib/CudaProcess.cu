#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "CudaProcess.cuh"
#include "Data.h"

//�z�X�g�������̃|�C���^
CudaInputMaster* cu_HostMaster;
int* cu_HostResultCount;
_u64* cu_HostResult;

//�f�o�C�X�������̃|�C���^
static CudaInputMaster* pDeviceMaster;
static int* pDeviceResultCount;
static _u64* pDeviceResult;

// ������s�萔
const int c_SizeBlockX = 1024;
//const int c_SizeBlockX = 1;
const int c_SizeBlockY = 1;
const int c_SizeGridX = 1024 * 512;
const int c_SizeGridY = 1;
//const int c_SizeGrid = 1;
const int c_SizeResult = 32;

// GPU�R�[�h
__device__ inline _u32 GetSignature(_u32 value)
{
	value ^= (value >> 16);
	value ^= (value >>  8);
	value ^= (value >>  4);
	value ^= (value >>  2);
	return (value ^ (value >> 1)) & 1;
}
__device__ inline _u32 Next(_u32* seeds, _u32 mask)
{
	_u32 value = (seeds[1] + seeds[3]) & mask;

	// m_S1 = m_S0 ^ m_S1;
	seeds[4] = seeds[0] ^ seeds[2];
	seeds[5] = seeds[1] ^ seeds[3];

	// m_S1 = RotateLeft(m_S1, 37);
	seeds[2] = seeds[5] << 5 | seeds[4] >> 27;
	seeds[3] = seeds[4] << 5 | seeds[5] >> 27;

	// m_S0 = RotateLeft(m_S0, 24) ^ m_S1 ^ (m_S1 << 16)
	seeds[6] = (seeds[0] << 24 | seeds[1] >> 8) ^ seeds[4] ^ (seeds[4] << 16 | seeds[5] >> 16);
	seeds[1] = (seeds[1] << 24 | seeds[0] >> 8) ^ seeds[5] ^ (seeds[5] << 16);

	seeds[0] = seeds[6];

	return value;
}
__device__ inline void Next(_u32* seeds)
{
	// m_S1 = m_S0 ^ m_S1;
	seeds[4] = seeds[0] ^ seeds[2];
	seeds[5] = seeds[1] ^ seeds[3];

	// m_S1 = RotateLeft(m_S1, 37);
	seeds[2] = seeds[5] << 5 | seeds[4] >> 27;
	seeds[3] = seeds[4] << 5 | seeds[5] >> 27;

	// m_S0 = RotateLeft(m_S0, 24) ^ m_S1 ^ (m_S1 << 16)
	seeds[6] = (seeds[0] << 24 | seeds[1] >> 8) ^ seeds[4] ^ (seeds[4] << 16 | seeds[5] >> 16);
	seeds[1] = (seeds[1] << 24 | seeds[0] >> 8) ^ seeds[5] ^ (seeds[5] << 16);

	seeds[0] = seeds[6];
}

// �v�Z����J�[�l��
__global__ void kernel_calc(CudaInputMaster* pSrc, int* pResultCount, _u64 *pResult, _u32 ivs)
{
//	int idx = blockDim.x * blockIdx.x + threadIdx.x; //�����̃X���b�hx��index
//	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int targetId = (blockIdx.x / 16) * 1024 + threadIdx.x;
	int chunkId = blockIdx.x % 16;

	ivs |= targetId;

	_u32 targetUpper = 0;
	_u32 targetLower = 0;

	// ����25bit = �̒l
	targetUpper |= (ivs &  0x1F00000ul); // iv0_0
	targetLower |= ((ivs &     0x3E0ul) << 10); // iv3_0
	targetUpper |= ((ivs &   0xF8000ul) >> 5); // iv1_0
	targetLower |= ((ivs &      0x1Ful) << 5); // iv4_0
	targetUpper |= ((ivs &    0x7C00ul) >> 10); // iv2_0

	// �B���ꂽ�l�𐄒�
	targetUpper |= ((32ul + pSrc->ivs[0] - ((ivs & 0x1F00000ul) >> 20)) & 0x1F) << 15;
	targetLower |= ((32ul + pSrc->ivs[3] - ((ivs &     0x3E0ul) >> 5))  & 0x1F) << 10;
	targetUpper |= ((32ul + pSrc->ivs[1] - ((ivs &   0xF8000ul) >> 15)) & 0x1F) <<  5;
	targetLower |= ((32ul + pSrc->ivs[4] - (ivs &      0x1Ful))         & 0x1F);
	targetLower |= ((32ul + pSrc->ivs[2] - ((ivs &    0x7C00ul) >> 10)) & 0x1F) << 20;
//	targetLower |= ((32ul + pSrc->ivs[5] - (ivs &        0x1Ful)) & 0x1F);
//	targetLower |= ((32ul + idy - (ivs &        0x1Ful)) & 0x1F);

	// target�x�N�g�����͊���

	targetUpper ^= pSrc->constantTermVector[0];
	targetLower ^= pSrc->constantTermVector[1];

	// ���������L���b�V��

	__shared__ _u32 answerFlag[128];
	__shared__ _u32 coefficientData[1024 * 2];
	__shared__ _u32 searchPattern[1024];
	__shared__ PokemonData pokemon[4];
	__shared__ int ecBit;
	__shared__ bool ecMod[3][6];

	if(threadIdx.x % 8 == 0)
	{
		answerFlag[threadIdx.x / 8] = pSrc->answerFlag[threadIdx.x / 8];
	}
	else if(threadIdx.x % 8 == 1)
	{
		pokemon[0] = pSrc->pokemon[0];
	}
	else if(threadIdx.x % 8 == 2)
	{
		pokemon[1] = pSrc->pokemon[1];
	}
	else if(threadIdx.x % 8 == 3)
	{
		pokemon[2] = pSrc->pokemon[2];
	}
	else if(threadIdx.x % 8 == 4)
	{
		pokemon[3] = pSrc->pokemon[3];
	}
	else if(threadIdx.x % 8 == 5)
	{
		ecBit = pSrc->ecBit;
	}
	else if(threadIdx.x % 8 == 6)
	{
		ecMod[0][0] = pSrc->ecMod[0][0];
		ecMod[0][1] = pSrc->ecMod[0][1];
		ecMod[0][2] = pSrc->ecMod[0][2];
		ecMod[0][3] = pSrc->ecMod[0][3];
		ecMod[0][4] = pSrc->ecMod[0][4];
		ecMod[0][5] = pSrc->ecMod[0][5];
		ecMod[1][0] = pSrc->ecMod[1][0];
		ecMod[1][1] = pSrc->ecMod[1][1];
		ecMod[1][2] = pSrc->ecMod[1][2];
	}
	else if(threadIdx.x % 8 == 7)
	{
		ecMod[1][3] = pSrc->ecMod[1][3];
		ecMod[1][4] = pSrc->ecMod[1][4];
		ecMod[1][5] = pSrc->ecMod[1][5];
		ecMod[2][0] = pSrc->ecMod[2][0];
		ecMod[2][1] = pSrc->ecMod[2][1];
		ecMod[2][2] = pSrc->ecMod[2][2];
		ecMod[2][3] = pSrc->ecMod[2][3];
		ecMod[2][4] = pSrc->ecMod[2][4];
		ecMod[2][5] = pSrc->ecMod[2][5];
	}
	coefficientData[threadIdx.x * 2]     = pSrc->coefficientData[chunkId * 2048 + threadIdx.x * 2];
	coefficientData[threadIdx.x * 2 + 1] = pSrc->coefficientData[chunkId * 2048 + threadIdx.x * 2 + 1];
	searchPattern[threadIdx.x] = pSrc->searchPattern[chunkId * 1024 + threadIdx.x];

	__syncthreads();

	_u32 processedTargetUpper = 0;
	_u32 processedTargetLower = 0;
	for(int i = 0; i < 32; ++i)
	{
		processedTargetUpper |= (GetSignature(answerFlag[i * 2] & targetUpper) ^ GetSignature(answerFlag[i * 2 + 1] & targetLower)) << (31 - i);
		processedTargetLower |= (GetSignature(answerFlag[(i + 32) * 2] & targetUpper) ^ GetSignature(answerFlag[(i + 32) * 2 + 1] & targetLower)) << (31 - i);
	}

	_u32 seeds[7]; // S0Upper�AS0Lower�AS1Upper�AS1Lower
	_u32 next[7]; // S0Upper�AS0Lower�AS1Upper�AS1Lower
	_u64 temp64;
	_u32 temp32;
	for(int i = 0; i < 1024; ++i)
	{
		seeds[0] = processedTargetUpper ^ coefficientData[i * 2];
		seeds[1] = processedTargetLower ^ coefficientData[i * 2 + 1] | searchPattern[i];

		// ��`�ӏ�

		if(ecBit >= 0 && (seeds[1] & 1) != ecBit)
		{
			continue;
		}

		temp64 = ((_u64)seeds[0] << 32 | seeds[1]) + 0x82a2b175229d6a5bull;

		seeds[2] = 0x82a2b175ul;
		seeds[3] = 0x229d6a5bul;

		next[0] = (_u32)(temp64 >> 32);
		next[1] = (_u32)temp64;
		next[2] = 0x82a2b175ul;
		next[3] = 0x229d6a5bul;

		temp64 = ((_u64)seeds[0] << 32 | seeds[1]);

		// ��������i�荞��

		// EC
		temp32 = Next(seeds, 0xFFFFFFFFu);
		// 1�C�ڌ�
		if(ecMod[0][temp32 % 6] == false)
		{
			continue;
		}
		// 2�C�ڌ�
		if(ecMod[1][temp32 % 6] == false)
		{
			continue;
		}

		// EC
		temp32 = Next(next, 0xFFFFFFFFu);
		// 3�C�ڌ�
		if(ecMod[2][temp32 % 6] == false)
		{
			continue;
		}

		// 2�C�ڂ��Ƀ`�F�b�N
		Next(next); // OTID
		Next(next); // PID

		{
			int ivs[6] = { -1, -1, -1, -1, -1, -1 };
			temp32 = 0;
			do {
				int fixedIndex = 0;
				do {
					fixedIndex = Next(next, 7); // V�ӏ�
				} while(fixedIndex >= 6);

				if(ivs[fixedIndex] == -1)
				{
					ivs[fixedIndex] = 31;
					++temp32;
				}
			} while(temp32 < pokemon[2].flawlessIvs);

			// �̒l
			temp32 = 1;
			for(int i = 0; i < 6; ++i)
			{
				if(ivs[i] == 31)
				{
					if(pokemon[2].ivs[i] != 31)
					{
						temp32 = 0;
						break;
					}
				}
				else if(pokemon[2].ivs[i] != Next(next, 0x1F))
				{
					temp32 = 0;
					break;
				}
			}
			if(temp32 == 0)
			{
				continue;
			}
			
			// ����
			temp32 = 0;
			if(pokemon[2].abilityFlag == 3)
			{
				temp32 = Next(next, 1);
			}
			else
			{
				do {
					temp32 = Next(next, 3);
				} while(temp32 >= 3);
			}
			if((pokemon[2].ability >= 0 && pokemon[2].ability != temp32) || (pokemon[2].ability == -1 && temp32 >= 2))
			{
				continue;
			}

			// ���ʒl
			if(!pokemon[2].isNoGender)
			{
				temp32 = 0;
				do {
					temp32 = Next(next, 0xFF);
				} while(temp32 >= 253);
			}

			// ���i
			temp32 = 0;
			do {
				temp32 = Next(next, 0x1F);
			} while(temp32 >= 25);

			if(temp32 != pokemon[2].nature)
			{
				continue;
			}
		}

		// 1�C��
		Next(seeds); // OTID
		Next(seeds); // PIT

		{
			// ��Ԃ�ۑ�
			next[0] = seeds[0];
			next[1] = seeds[1];
			next[2] = seeds[2];
			next[3] = seeds[3];

			{
				int ivs[6] = { -1, -1, -1, -1, -1, -1 };
				temp32 = 0;
				do {
					int fixedIndex = 0;
					do {
						fixedIndex = Next(seeds, 7); // V�ӏ�
					} while(fixedIndex >= 6);

					if(ivs[fixedIndex] == -1)
					{
						ivs[fixedIndex] = 31;
						++temp32;
					}
				} while(temp32 < pokemon[0].flawlessIvs);

				// �̒l
				temp32 = 1;
				for(int i = 0; i < 6; ++i)
				{
					if(ivs[i] == 31)
					{
						if(pokemon[0].ivs[i] != 31)
						{
							temp32 = 0;
							break;
						}
					}
					else if(pokemon[0].ivs[i] != Next(seeds, 0x1F))
					{
						temp32 = 0;
						break;
					}
				}
				if(temp32 == 0)
				{
					continue;
				}
			}
			{
				int ivs[6] = { -1, -1, -1, -1, -1, -1 };
				temp32 = 0;
				do {
					int fixedIndex = 0;
					do {
						fixedIndex = Next(next, 7); // V�ӏ�
					} while(fixedIndex >= 6);

					if(ivs[fixedIndex] == -1)
					{
						ivs[fixedIndex] = 31;
						++temp32;
					}
				} while(temp32 < pokemon[1].flawlessIvs);

				// �̒l
				temp32 = 1;
				for(int i = 0; i < 6; ++i)
				{
					if(ivs[i] == 31)
					{
						if(pokemon[1].ivs[i] != 31)
						{
							temp32 = 0;
							break;
						}
					}
					else if(pokemon[1].ivs[i] != Next(next, 0x1F))
					{
						temp32 = 0;
						break;
					}
				}
				if(temp32 == 0)
				{
					continue;
				}
			}

			// ����
			temp32 = 0;
			if(pokemon[0].abilityFlag == 3)
			{
				temp32 = Next(seeds, 1);
			}
			else
			{
				do {
					temp32 = Next(seeds, 3);
				} while(temp32 >= 3);
			}
			if((pokemon[0].ability >= 0 && pokemon[0].ability != temp32) || (pokemon[0].ability == -1 && temp32 >= 2))
			{
				continue;
			}
			temp32 = 0;
			if(pokemon[1].abilityFlag == 3)
			{
				temp32 = Next(next, 1);
			}
			else
			{
				do {
					temp32 = Next(next, 3);
				} while(temp32 >= 3);
			}
			if((pokemon[1].ability >= 0 && pokemon[1].ability != temp32) || (pokemon[1].ability == -1 && temp32 >= 2))
			{
				continue;
			}

			// ���ʒl
			if(!pokemon[0].isNoGender)
			{
				temp32 = 0;
				do {
					temp32 = Next(seeds, 0xFF);
				} while(temp32 >= 253);
			}
			if(!pokemon[1].isNoGender)
			{
				temp32 = 0;
				do {
					temp32 = Next(next, 0xFF);
				} while(temp32 >= 253);
			}

			// ���i
			temp32 = 0;
			do {
				temp32 = Next(seeds, 0x1F);
			} while(temp32 >= 25);
			if(temp32 != pokemon[0].nature)
			{
				continue;
			}
			temp32 = 0;
			do {
				temp32 = Next(next, 0x1F);
			} while(temp32 >= 25);
			if(temp32 != pokemon[1].nature)
			{
				continue;
			}
		}
		// ���ʂ���������
		int old = atomicAdd(pResultCount, 1);
		pResult[old] = temp64;
	}
	return;
}

// ������
void CudaInitializeImpl()
{
	// �z�X�g�������̊m��
	hipHostMalloc(&cu_HostMaster, sizeof(CudaInputMaster));
	hipHostMalloc(&cu_HostResult, sizeof(_u64) * c_SizeResult);
	hipHostMalloc(&cu_HostResultCount, sizeof(int));

	// �f�[�^�̏�����
	cu_HostMaster->ecBit = -1;

	// �f�o�C�X�������̊m��
	hipMalloc(&pDeviceMaster, sizeof(CudaInputMaster));
	hipMalloc(&pDeviceResult, sizeof(_u64) * c_SizeResult);
	hipMalloc(&pDeviceResultCount, sizeof(int));
}

// �f�[�^�Z�b�g
void CudaSetMasterData(int length)
{
	cu_HostMaster->constantTermVector[0] = (_u32)(g_ConstantTermVector >> 25);
	cu_HostMaster->constantTermVector[1] = (_u32)(g_ConstantTermVector & 0x1FFFFFFull);
//	cu_HostMaster->constantTermVector[0] = (_u32)(g_ConstantTermVector >> (length / 2));
//	cu_HostMaster->constantTermVector[1] = (_u32)(g_ConstantTermVector & (1 << (length / 2 + 1) - 1));
	for(int i = 0; i < 64; ++i)
	{
		cu_HostMaster->answerFlag[i * 2] = (_u32)(g_AnswerFlag[i] >> 25);
		cu_HostMaster->answerFlag[i * 2 + 1] = (_u32)(g_AnswerFlag[i] & 0x1FFFFFFull);
	}
	for(int i = 0; i < 16 * 1024; ++i)
	{
		cu_HostMaster->coefficientData[i * 2] = (_u32)(g_CoefficientData[i] >> 32);
		cu_HostMaster->coefficientData[i * 2 + 1] = (_u32)(g_CoefficientData[i] & 0xFFFFFFFFull);
		cu_HostMaster->searchPattern[i] = (_u32)g_SearchPattern[i];
	}
	*cu_HostResultCount = 0;

	// �f�[�^��]��
	hipMemcpy(pDeviceMaster, cu_HostMaster, sizeof(CudaInputMaster), hipMemcpyHostToDevice);
	hipMemcpy(pDeviceResultCount, cu_HostResultCount, sizeof(int), hipMemcpyHostToDevice);
}

// �v�Z
void CudaProcess(_u32 ivs, int freeBit)
{
	//�J�[�l��
	dim3 block(c_SizeBlockX, c_SizeBlockY, 1);
	dim3 grid(c_SizeGridX, c_SizeGridY, 1);
	kernel_calc << < grid, block >> > (pDeviceMaster, pDeviceResultCount, pDeviceResult, ivs);

	//�f�o�C�X->�z�X�g�֌��ʂ�]��
	hipMemcpy(cu_HostResult, pDeviceResult, sizeof(_u64) * c_SizeResult, hipMemcpyDeviceToHost);
	hipMemcpy(cu_HostResultCount, pDeviceResultCount, sizeof(int), hipMemcpyDeviceToHost);
}

void Finish()
{
	//�f�o�C�X�������̊J��
	hipFree(pDeviceResultCount);
	hipFree(pDeviceResult);
	hipFree(pDeviceMaster);
	//�z�X�g�������̊J��
	hipHostFree(cu_HostResultCount);
	hipHostFree(cu_HostResult);
	hipHostFree(cu_HostMaster);
}
