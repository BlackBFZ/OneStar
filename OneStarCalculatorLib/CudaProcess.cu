#include "CudaProcess.cuh"

// �萔
static CudaConst* cu_HostConstData;
CudaConst* cu_DeviceConstData;

// �ϐ�����
CudaInputMaster* cu_HostInputMaster;
_u32* cu_HostInputCoefficientData;
_u32* cu_HostInputSearchPattern;

// ���ʋ���
int* cu_HostResultCount;
_u64* cu_HostResult;

// �萔
const int c_SizeResult = 32;

// ������
void CudaInitializeImpl()
{
	// �z�X�g�������̊m��
	hipHostMalloc(&cu_HostConstData, sizeof(CudaConst));
	hipHostMalloc(&cu_HostInputMaster, sizeof(CudaInputMaster));
	hipHostMalloc(&cu_HostResultCount, sizeof(int));
	hipHostMalloc(&cu_HostResult, sizeof(_u64) * c_SizeResult);

	// �f�o�C�X�������̊m��
	hipMalloc(&cu_DeviceConstData, sizeof(CudaConst));

	// �f�[�^�̏�����
	cu_HostInputMaster->ecBit = -1;

	// �萔�f�[�^��]��
//	cu_HostConstData->natureTable[0].randMax = 0x1F;
//	cu_HostConstData->natureTable[0].randMax = 0x1F;
	hipMemcpy(cu_DeviceConstData, cu_HostConstData, sizeof(CudaConst), hipMemcpyHostToDevice);
}

// �I��
void CudaFinalizeImpl()
{
	// �f�o�C�X���������
	hipFree(cu_DeviceConstData);

	// �z�X�g���������
	hipHostFree(cu_HostResult);
	hipHostFree(cu_HostResultCount);
	hipHostFree(cu_HostInputMaster);
	hipHostFree(cu_HostConstData);
}
