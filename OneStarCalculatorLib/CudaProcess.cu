#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "CudaProcess.cuh"
#include "Data.h"

//�z�X�g�������̃|�C���^
CudaInputMaster* pHostMaster; // �Œ�f�[�^
_u64* cu_HostResult;

//�f�o�C�X�������̃|�C���^
CudaInputMaster* pDeviceMaster;
_u64* pDeviceResult;

__device__ inline _u64 GetSignature(_u64 value)
{
	unsigned int a = (unsigned int)(value ^ (value >> 32));
	a = a ^ (a >> 16);
	a = a ^ (a >> 8);
	a = a ^ (a >> 4);
	a = a ^ (a >> 2);
	return (a ^ (a >> 1)) & 1;
}

// �v�Z����J�[�l��
__global__ void kernel_calc(CudaInputMaster* pSrc, _u64 *pResult, _u64 ivs)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x; //�����̃X���b�hx��index

	ivs |= idx;

	_u64 target = 0;

	// ����30bit = �̒l
	target |= (ivs & 0x3E000000ul) << 30; // iv0_0
	target |= (ivs & 0x1F00000ul) << 25; // iv1_0
	target |= (ivs & 0xF8000ul) << 20; // iv2_0
	target |= (ivs & 0x7C00ul) << 15; // iv3_0
	target |= (ivs & 0x3E0ul) << 10; // iv4_0
	target |= (ivs & 0x1Ful) << 5; // iv5_0

	// �B���ꂽ�l�𐄒�
	target |= ((32ul + pSrc->ivs[0] - ((ivs & 0x3E000000ul) >> 25)) & 0x1F) << 50;
	target |= ((32ul + pSrc->ivs[1] - ((ivs & 0x1F00000ul) >> 20)) & 0x1F) << 40;
	target |= ((32ul + pSrc->ivs[2] - ((ivs & 0xF8000ul) >> 15)) & 0x1F) << 30;
	target |= ((32ul + pSrc->ivs[3] - ((ivs & 0x7C00ul) >> 10)) & 0x1F) << 20;
	target |= ((32ul + pSrc->ivs[4] - ((ivs & 0x3E0ul) >> 5)) & 0x1F) << 10;
	target |= ((32ul + pSrc->ivs[5] - (ivs & 0x1Ful)) & 0x1F);

	// target�x�N�g�����͊���

	target ^= pSrc->constantTermVector;
	// 60bit���̌v�Z���ʃL���b�V��

	_u64 processedTarget = 0;
	_u64 v;
//	unsigned int a;
	for(int i = 0; i < 60; ++i)
	{
		processedTarget |= (GetSignature(pSrc->answerFlag[i] & target) << (63 - i));
		/*
		v = pSrc->answerFlag[i] & target;
		v = (v ^ (v >> 32));
		v = v ^ (v >> 16);
		v = v ^ (v >> 8);
		v = v ^ (v >> 4);
		v = v ^ (v >> 2);
		processedTarget |= ((v ^ (v >> 1)) & 1) << (63 - i);
		*/
	}

	pResult[idx] = processedTarget;
	return;
}

// ������
void CudaInitialize(int* pIvs)
{
	// �z�X�g�������̊m��
	hipHostMalloc(&pHostMaster, sizeof(CudaInputMaster));
	hipHostMalloc(&cu_HostResult, sizeof(_u64) * 1024 * 1024 * 16);

	// �f�o�C�X�������̊m��
	hipMalloc(&pDeviceMaster, sizeof(CudaInputMaster));
	hipMalloc(&pDeviceResult, sizeof(_u64) * 1024 * 1024 * 16);

	// �}�X�^�[�f�[�^�̃Z�b�g
	for(int i = 0; i < 6; ++i)
	{
		pHostMaster->ivs[i] = pIvs[i];
	}
	pHostMaster->constantTermVector = g_ConstantTermVector;
	for(int i = 0; i < 64; ++i)
	{
		pHostMaster->answerFlag[i] = g_AnswerFlag[i];
	}

	// �f�[�^��]��
	hipMemcpy(pDeviceMaster, pHostMaster, sizeof(CudaInputMaster), hipMemcpyHostToDevice);
}

// �v�Z
void CudaProcess(_u64 ivs, int freeBit)
{
	//�J�[�l��
	dim3 block(1024, 1, 1);
	dim3 grid(1024*16, 1, 1);
	kernel_calc << < grid, block >> > (pDeviceMaster, pDeviceResult, ivs);

	//�f�o�C�X->�z�X�g�֌��ʂ�]��
	hipMemcpy(cu_HostResult, pDeviceResult, sizeof(_u64) * 1024 * 1024 * 16, hipMemcpyDeviceToHost);
}

void Finish()
{
	//�f�o�C�X�������̊J��
	hipFree(pDeviceMaster);
	hipFree(pDeviceResult);
	//�z�X�g�������̊J��
	hipHostFree(pHostMaster);
	hipHostFree(cu_HostResult);
}
