#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "CudaProcess.cuh"
#include "Data.h"

//�z�X�g�������̃|�C���^
CudaInputMaster* cu_HostMaster;
_u64* cu_HostResult;

//�f�o�C�X�������̃|�C���^
static CudaInputMaster* pDeviceMaster;
static _u64* pDeviceResult;

// ������s�萔
const int c_SizeBlockX = 64;
const int c_SizeBlockY = 16;
const int c_SizeGrid = 1024 * 16;
const int c_SizeResult = 16;

// GPU�R�[�h
__device__ inline _u32 GetSignature(_u32 value)
{
	value ^= (value >> 16);
	value ^= (value >>  8);
	value ^= (value >>  4);
	value ^= (value >>  2);
	return (value ^ (value >> 1)) & 1;
}
__device__ inline _u32 Next(_u32* seeds, _u32 mask)
{
	_u32 value = (seeds[1] + seeds[3]) & mask;

	// m_S1 = m_S0 ^ m_S1;
	seeds[4] = seeds[0] ^ seeds[2];
	seeds[5] = seeds[1] ^ seeds[3];

	// m_S1 = RotateLeft(m_S1, 37);
	seeds[2] = seeds[5] << 5 | seeds[4] >> 27;
	seeds[3] = seeds[4] << 5 | seeds[5] >> 27;

	// m_S0 = RotateLeft(m_S0, 24) ^ m_S1 ^ (m_S1 << 16)
	seeds[6] = (seeds[0] << 24 | seeds[1] >> 8) ^ seeds[4] ^ (seeds[4] << 16 | seeds[5] >> 16);
	seeds[1] = (seeds[1] << 24 | seeds[0] >> 8) ^ seeds[5] ^ (seeds[5] << 16);

	seeds[0] = seeds[6];

	return value;
}
__device__ inline void Next(_u32* seeds)
{
	// m_S1 = m_S0 ^ m_S1;
	seeds[4] = seeds[0] ^ seeds[2];
	seeds[5] = seeds[1] ^ seeds[3];

	// m_S1 = RotateLeft(m_S1, 37);
	seeds[2] = seeds[5] << 5 | seeds[4] >> 27;
	seeds[3] = seeds[4] << 5 | seeds[5] >> 27;

	// m_S0 = RotateLeft(m_S0, 24) ^ m_S1 ^ (m_S1 << 16)
	seeds[6] = (seeds[0] << 24 | seeds[1] >> 8) ^ seeds[4] ^ (seeds[4] << 16 | seeds[5] >> 16);
	seeds[1] = (seeds[1] << 24 | seeds[0] >> 8) ^ seeds[5] ^ (seeds[5] << 16);

	seeds[0] = seeds[6];
}

// �v�Z����J�[�l��
__global__ void kernel_calc(CudaInputMaster* pSrc, _u64 *pResult, _u32 ivs)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x; //�����̃X���b�hx��index
	int idy = blockDim.y * blockIdx.y + threadIdx.y;

	ivs |= idx;

	_u32 targetUpper = 0;
	_u32 targetLower = 0;

	// ����30bit = �̒l
	targetUpper |= (ivs & 0x3E000000ul); // iv0_0
	targetLower |= ((ivs &    0x7C00ul) << 15); // iv3_0
	targetUpper |= ((ivs & 0x1F00000ul) >> 5); // iv1_0
	targetLower |= ((ivs &     0x3E0ul) << 10); // iv4_0
	targetUpper |= ((ivs &   0xF8000ul) >> 10); // iv2_0
	targetLower |= ((ivs &      0x1Ful) << 5); // iv5_0

	// �B���ꂽ�l�𐄒�
	targetUpper |= ((32ul + pSrc->ivs[0] - ((ivs & 0x3E000000ul) >> 25)) & 0x1F) << 20;
	targetLower |= ((32ul + pSrc->ivs[3] - ((ivs &     0x7C00ul) >> 10)) & 0x1F) << 20;
	targetUpper |= ((32ul + pSrc->ivs[1] - ((ivs &  0x1F00000ul) >> 20)) & 0x1F) << 10;
	targetLower |= ((32ul + pSrc->ivs[4] - ((ivs &      0x3E0ul) >> 5)) & 0x1F) << 10;
	targetUpper |= ((32ul + pSrc->ivs[2] - ((ivs &    0xF8000ul) >> 15)) & 0x1F);
	targetLower |= ((32ul + pSrc->ivs[5] - (ivs &        0x1Ful)) & 0x1F);
//	targetLower |= ((32ul + idy - (ivs &        0x1Ful)) & 0x1F);

	// target�x�N�g�����͊���

	targetUpper ^= pSrc->constantTermVector[0];
	targetLower ^= pSrc->constantTermVector[1];

	// 60bit���̌v�Z���ʃL���b�V��

	__shared__ _u32 processedTargetUpper[64];
	__shared__ _u32 processedTargetLower[64];

//	_u32 processedTargetUpper = 0;
//	_u32 processedTargetLower = 0;
	processedTargetUpper[threadIdx.x] = 0;
	processedTargetLower[threadIdx.x] = 0;
	for(int i = 0; i < 32; ++i)
	{
		processedTargetUpper[threadIdx.x] |= (GetSignature(pSrc->answerFlag[i * 2] & targetUpper) ^ GetSignature(pSrc->answerFlag[i * 2 + 1] & targetLower)) << (31 - i);
		processedTargetLower[threadIdx.x] |= (GetSignature(pSrc->answerFlag[(i + 32) * 2] & targetUpper) ^ GetSignature(pSrc->answerFlag[(i + 32) * 2 + 1] & targetLower)) << (31 - i);
	}

	// �X���b�h�𓯊�
	__syncthreads();

	_u32 seeds[7]; // S0Upper�AS0Lower�AS1Upper�AS1Lower
	_u32 next[7]; // S0Upper�AS0Lower�AS1Upper�AS1Lower
	_u64 temp64;
	_u32 temp32;
//	for(int i = 0; i < 16; ++i)
	{
		seeds[0] = processedTargetUpper[threadIdx.x] ^ pSrc->coefficientData[idy * 2];
		seeds[1] = processedTargetLower[threadIdx.x] ^ pSrc->coefficientData[idy * 2 + 1] | pSrc->searchPattern[idy];

		// ��`�ӏ�

		if(pSrc->ecBit >= 0 && (seeds[1] & 1) != pSrc->ecBit)
		{
			return;
		}

		temp64 = ((_u64)seeds[0] << 32 | seeds[1]) + 0x82a2b175229d6a5bull;

		seeds[2] = 0x82a2b175ul;
		seeds[3] = 0x229d6a5bul;

		next[0] = (_u32)(temp64 >> 32);
		next[1] = (_u32)temp64;
		next[2] = 0x82a2b175ul;
		next[3] = 0x229d6a5bul;

		temp64 = ((_u64)seeds[0] << 32 | seeds[1]);

		// ��������i�荞��

		// EC
		temp32 = Next(seeds, 0xFFFFFFFFu);
		// 1�C�ڌ�
		if(pSrc->ecMod[0][temp32 % 6] == false)
		{
			return;
		}
		// 2�C�ڌ�
		if(pSrc->ecMod[1][temp32 % 6] == false)
		{
			return;
		}

		// EC
		temp32 = Next(next, 0xFFFFFFFFu);
		// 3�C�ڌ�
		if(pSrc->ecMod[2][temp32 % 6] == false)
		{
			return;
		}

		// 2�C�ڂ��Ƀ`�F�b�N
		Next(next); // OTID
		Next(next); // PID

		{
			int ivs[6] = { -1, -1, -1, -1, -1, -1 };
			temp32 = 0;
			do {
				int fixedIndex = 0;
				do {
					fixedIndex = Next(next, 7); // V�ӏ�
				} while(fixedIndex >= 6);

				if(ivs[fixedIndex] == -1)
				{
					ivs[fixedIndex] = 31;
					++temp32;
				}
			} while(temp32 < pSrc->pokemon[2].flawlessIvs);

			// �̒l
			temp32 = 1;
			for(int i = 0; i < 6; ++i)
			{
				if(ivs[i] == 31)
				{
					if(pSrc->pokemon[2].ivs[i] != 31)
					{
						temp32 = 0;
						break;
					}
				}
				else if(pSrc->pokemon[2].ivs[i] != Next(next, 0x1F))
				{
					temp32 = 0;
					break;
				}
			}
			if(temp32 == 0)
			{
				return;
			}
			
			// ����
			temp32 = 0;
			if(pSrc->pokemon[2].abilityFlag == 3)
			{
				temp32 = Next(next, 1);
			}
			else
			{
				do {
					temp32 = Next(next, 3);
				} while(temp32 >= 3);
			}
			if((pSrc->pokemon[2].ability >= 0 && pSrc->pokemon[2].ability != temp32) || (pSrc->pokemon[2].ability == -1 && temp32 >= 2))
			{
				return;
			}

			// ���ʒl
			if(!pSrc->pokemon[2].isNoGender)
			{
				temp32 = 0;
				do {
					temp32 = Next(next, 0xFF);
				} while(temp32 >= 253);
			}

			// ���i
			temp32 = 0;
			do {
				temp32 = Next(next, 0x1F);
			} while(temp32 >= 25);

			if(temp32 != pSrc->pokemon[2].nature)
			{
				return;
			}
		}

		// 1�C��
		Next(seeds); // OTID
		Next(seeds); // PID

		{
			// ��Ԃ�ۑ�
			next[0] = seeds[0];
			next[1] = seeds[1];
			next[2] = seeds[2];
			next[3] = seeds[3];

			{
				int ivs[6] = { -1, -1, -1, -1, -1, -1 };
				temp32 = 0;
				do {
					int fixedIndex = 0;
					do {
						fixedIndex = Next(seeds, 7); // V�ӏ�
					} while(fixedIndex >= 6);

					if(ivs[fixedIndex] == -1)
					{
						ivs[fixedIndex] = 31;
						++temp32;
					}
				} while(temp32 < pSrc->pokemon[0].flawlessIvs);

				// �̒l
				temp32 = 1;
				for(int i = 0; i < 6; ++i)
				{
					if(ivs[i] == 31)
					{
						if(pSrc->pokemon[0].ivs[i] != 31)
						{
							temp32 = 0;
							break;
						}
					}
					else if(pSrc->pokemon[0].ivs[i] != Next(seeds, 0x1F))
					{
						temp32 = 0;
						break;
					}
				}
				if(temp32 == 0)
				{
					return;
				}
			}
			{
				int ivs[6] = { -1, -1, -1, -1, -1, -1 };
				temp32 = 0;
				do {
					int fixedIndex = 0;
					do {
						fixedIndex = Next(next, 7); // V�ӏ�
					} while(fixedIndex >= 6);

					if(ivs[fixedIndex] == -1)
					{
						ivs[fixedIndex] = 31;
						++temp32;
					}
				} while(temp32 < pSrc->pokemon[1].flawlessIvs);

				// �̒l
				temp32 = 1;
				for(int i = 0; i < 6; ++i)
				{
					if(ivs[i] == 31)
					{
						if(pSrc->pokemon[1].ivs[i] != 31)
						{
							temp32 = 0;
							break;
						}
					}
					else if(pSrc->pokemon[1].ivs[i] != Next(next, 0x1F))
					{
						temp32 = 0;
						break;
					}
				}
				if(temp32 == 0)
				{
					return;
				}
			}

			// ����
			temp32 = 0;
			if(pSrc->pokemon[0].abilityFlag == 3)
			{
				temp32 = Next(seeds, 1);
			}
			else
			{
				do {
					temp32 = Next(seeds, 3);
				} while(temp32 >= 3);
			}
			if((pSrc->pokemon[0].ability >= 0 && pSrc->pokemon[0].ability != temp32) || (pSrc->pokemon[0].ability == -1 && temp32 >= 2))
			{
				return;
			}
			temp32 = 0;
			if(pSrc->pokemon[1].abilityFlag == 3)
			{
				temp32 = Next(next, 1);
			}
			else
			{
				do {
					temp32 = Next(next, 3);
				} while(temp32 >= 3);
			}
			if((pSrc->pokemon[1].ability >= 0 && pSrc->pokemon[1].ability != temp32) || (pSrc->pokemon[1].ability == -1 && temp32 >= 2))
			{
				return;
			}

			// ���ʒl
			if(!pSrc->pokemon[0].isNoGender)
			{
				temp32 = 0;
				do {
					temp32 = Next(seeds, 0xFF);
				} while(temp32 >= 253);
			}
			if(!pSrc->pokemon[1].isNoGender)
			{
				temp32 = 0;
				do {
					temp32 = Next(next, 0xFF);
				} while(temp32 >= 253);
			}

			// ���i
			temp32 = 0;
			do {
				temp32 = Next(seeds, 0x1F);
			} while(temp32 >= 25);
			if(temp32 != pSrc->pokemon[0].nature)
			{
				return;
			}
			temp32 = 0;
			do {
				temp32 = Next(next, 0x1F);
			} while(temp32 >= 25);
			if(temp32 != pSrc->pokemon[1].nature)
			{
				return;
			}
		}

		pResult[0] = temp64;
	}
	return;
}

// ������
void CudaInitializeImpl()
{
	// �z�X�g�������̊m��
	hipHostMalloc(&cu_HostMaster, sizeof(CudaInputMaster));

	{
		auto errorCode = hipGetLastError();
		auto errorStr = hipGetErrorName(errorCode);
		;
	}

	hipHostMalloc(&cu_HostResult, sizeof(_u64) * c_SizeResult);

	{
		auto errorCode = hipGetLastError();
		auto errorStr = hipGetErrorName(errorCode);
		;
	}

	// �f�[�^�̏�����
	cu_HostMaster->ecBit = -1;

	// �f�o�C�X�������̊m��
	hipMalloc(&pDeviceMaster, sizeof(CudaInputMaster));

	{
		auto errorCode = hipGetLastError();
		auto errorStr = hipGetErrorName(errorCode);
		;
	}

	hipMalloc(&pDeviceResult, sizeof(_u64) * c_SizeResult);

	{
		auto errorCode = hipGetLastError();
		auto errorStr = hipGetErrorName(errorCode);
		;
	}

}

// �f�[�^�Z�b�g
void CudaSetMasterData()
{
	cu_HostMaster->constantTermVector[0] = (_u32)(g_ConstantTermVector >> 30);
	cu_HostMaster->constantTermVector[1] = (_u32)(g_ConstantTermVector & 0x3FFFFFFFull);
	for(int i = 0; i < 64; ++i)
	{
		cu_HostMaster->answerFlag[i * 2] = (_u32)(g_AnswerFlag[i] >> 30);
		cu_HostMaster->answerFlag[i * 2 + 1] = (_u32)(g_AnswerFlag[i] & 0x3FFFFFFFull);
	}
	for(int i = 0; i < 16; ++i)
	{
		cu_HostMaster->coefficientData[i * 2] = (_u32)(g_CoefficientData[i] >> 32);
		cu_HostMaster->coefficientData[i * 2 + 1] = (_u32)(g_CoefficientData[i] & 0xFFFFFFFFull);
		cu_HostMaster->searchPattern[i] = (_u32)g_SearchPattern[i];
	}

	// �f�[�^��]��
	hipMemcpy(pDeviceMaster, cu_HostMaster, sizeof(CudaInputMaster), hipMemcpyHostToDevice);
}

// �v�Z
void CudaProcess(_u32 ivs, int freeBit)
{
	//�J�[�l��
	dim3 block(c_SizeBlockX, c_SizeBlockY, 1);
	dim3 grid(c_SizeGrid, 1, 1);
	kernel_calc << < grid, block >> > (pDeviceMaster, pDeviceResult, ivs);

	auto errorCode = hipGetLastError();
	auto errorStr = hipGetErrorName(errorCode);

	//�f�o�C�X->�z�X�g�֌��ʂ�]��
	hipMemcpy(cu_HostResult, pDeviceResult, sizeof(_u64) * c_SizeResult, hipMemcpyDeviceToHost);
}

void Finish()
{
	//�f�o�C�X�������̊J��
	hipFree(pDeviceResult);
	hipFree(pDeviceMaster);
	//�z�X�g�������̊J��
	hipHostFree(cu_HostResult);
	hipHostFree(cu_HostMaster);
}
